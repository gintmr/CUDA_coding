
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello from GPU!\n");
}

 
int main() 
{
    hello_from_gpu<<<2,1>>>();
    hipDeviceSynchronize();
    printf("Jellyfish-mode\n");
    return 0;
}